
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * quite simple application to stress a GPU (runs for a couple seconds)
 *
 * compile with: 'nvcc -o stress stress.cu'
 */

__global__
void worker(int size, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < size)
  {
      x[i] = x[i] * y[i];
      x[i] = pow(y[i], x[i]);
      x[i] = log(x[i]);
      x[i] = sqrt(x[i]);
      x[i] = cos(x[i]);
      x[i] = sin(y[i]);
  }
}

int main()
{
    const int SIZE = 1024 * 1024;
    const int ITERATIONS = 1024 * 1024;

    float *x = (float *)malloc(SIZE * sizeof(float));
    float *y = (float *)malloc(SIZE * sizeof(float));

    for (int i = 0; i < SIZE; i++)
    {
        x[i] = 1.0;
        y[i] = (float)i / SIZE;
    }

    float *cuda_x;
    float *cuda_y;

    hipMalloc(&cuda_x, SIZE * sizeof(float));
    hipMalloc(&cuda_y, SIZE * sizeof(float));

    hipMemcpy(cuda_x, x, SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuda_y, y, SIZE * sizeof(float), hipMemcpyHostToDevice);

    for(int i = 0; i < ITERATIONS; i++)
        worker<<<(SIZE + 255) /  256, 256>>>(SIZE, cuda_x, cuda_y);

    hipFree(cuda_y);
    hipFree(cuda_x);
    free(y);
    free(x);

    return 0;
}
